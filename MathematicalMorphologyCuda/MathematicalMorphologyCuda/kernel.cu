#include "hip/hip_runtime.h"

#include "MathematicalMorphologyCuda.h"
#define BLOCK_2D 22
#define WHITE 255
#define BLACK 0

/*
*	Kernel used to split image channels
*		image: input image that has to be split
*		red: matrix of pixels for red channel
*		green: matrix of pixels for green channel
*		blue: matrix of pixels for blue channel
*		width: image width
*		height: image height
*		structWidth: width of structuring element
*		structHeight: height of structuring element
*		ghost: value for ghost cells
*/
__global__ void SplitChannels(uint8_t* image, uint8_t* red,
	uint8_t* green, uint8_t* blue, int width, int height, 
	int structWidth, int structHeight, int ghost)
{
	int j = 0;
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int firstX = (structWidth - 1) / 2;
	int firstY = (structHeight - 1) / 2;
	int lastX = width + firstX;
	int lastY = height + firstY;
	int sizeX = width + structWidth - 1;
	int sizeY = height + structHeight - 1;
	int img = x >= firstX && x < lastX &&
		y >= firstY && y < lastY;
	int halo = !img;
	int index = y * sizeX + x;
	int i = (y - firstY)*width + x - firstX;
	if (x < sizeX && y < sizeY)
	{
		blue[index] = image[i*CHANNELS + j] * img + ghost * halo;
		j++;
		green[index] = image[i*CHANNELS + j] * img + ghost * halo;
		j++;
		red[index] = image[i*CHANNELS + j] * img + ghost * halo;
	}
}

/*
*	Kernel that sets the default value for output channels
*		red: red channel
*		green: green channel
*		blue: blue channel
*		width: total channel width, considering also ghost cells
*		height: total channel height, considering also ghost cells
*		value: default value
*/
__global__ void setDefault(uint8_t* red, uint8_t* green, 
	uint8_t* blue, int width, int height, uint8_t value)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int index = y * width + x;
	if (x < width && y < height)
	{
		red[index] = value;
		green[index] = value;
		blue[index] = value;
	}
}

/*
*	Kernel that composes an image from its channels
*		image: output image
*		red: red channel
*		green: green channel
*		blue: blue channel
*		structWidth: width of structuring element
*		structHeight: height of structuring element
*		width: image width
*		height: image height
*/
__global__ void ComposeImage(uint8_t* image, uint8_t* red,
	uint8_t* green, uint8_t* blue, int structWidth, 
	int structHeight, int width, int height)
{
	int j = 0;
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int firstX = (structWidth - 1) / 2;
	int firstY = (structHeight - 1) / 2;
	int lastX = width + firstX;
	int lastY = height + firstY;
	int sizeW = width + structWidth - 1;
	int index = y * sizeW + x;
	int i = (y - firstY)*width + x - firstX;
	if (x >= firstX && x < lastX &&
		y >= firstY && y < lastY)
	{
		image[i*CHANNELS + j] = blue[index];
		j++;
		image[i*CHANNELS + j] = green[index];
		j++;
		image[i*CHANNELS + j] = red[index];
		j++;
		image[i*CHANNELS + j] = ALPHA;
	}
}

/*
*	Kernel that executes erosion operation
*		input: input channel
*		output: output channel
*		width: image width
*		height: image height
*		structWidth: width of structuring element
*		structHeight: height of structuring element
*		offset: array of offsets
*		offCount: number of element inside offset
*/
__global__ void Erosion(uint8_t* input, uint8_t* output,
	int width, int height, int structWidth, int structHeight,
	int* offset, int offCount)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int firstX = (structWidth - 1) / 2;
	int firstY = (structHeight - 1) / 2;
	int lastX = width + firstX;
	int lastY = height + firstY;
	int sizeW = width + structWidth - 1;
	int index = y * sizeW + x;
	uint8_t minValue = WHITE;
	if (x >= firstX && x < lastX &&
		y >= firstY && y < lastY)
	{
		for (int i = 0; i < offCount; i++)
		{
			if (input[index + offset[i]] < minValue)
			{
				minValue = input[index + offset[i]];
			}
		}
		output[index] = minValue;
	}
}

/*
*	Kernel that executes dilation operation
*		input: input channel
*		output: output channel
*		width: image width
*		height: image height
*		structWidth: width of structuring element
*		structHeight: height of structuring element
*		offset: array of offsets
*		offCount: number of element inside offset
*/
__global__ void Dilation(uint8_t* input, uint8_t* output,
	int width, int height, int structWidth, int structHeight,
	int* offset, int offCount)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int firstX = (structWidth - 1) / 2;
	int firstY = (structHeight - 1) / 2;
	int lastX = width + firstX;
	int lastY = height + firstY;
	int sizeW = width + structWidth - 1;
	int index = y * sizeW + x;
	uint8_t maxValue = BLACK;
	if (x >= firstX && x < lastX &&
		y >= firstY && y < lastY)
	{
		for (int i = 0; i < offCount; i++)
		{
			if (input[index + offset[i]] > maxValue)
			{
				maxValue = input[index + offset[i]];
			}
		}
		output[index] = maxValue;
	}
}

/*
*	Function that will be called by the Unreal Engine classes to 
*	execute opening or closing operations with CUDA
*		structWidth: width of structuring element
*		structHeight: height of structuring element
*		image: input image
*		width: image width
*		height: image height
*		erOffset: array of offsets for erosion operations
*		erCount: number of elements in erOffset
*		dilOffset: array of offsets for dilation operations
*		dilCount: number of elements in dilOffset
*		isOpening: true if we have to execute opening operations,
*			false otherwise
*/
uint8_t* CudaMathMorphology::ExecuteOpeningOrClosing(int structWidth,
	int structHeight, uint8_t* image, int width,
	int height, int* erOffset, int erCount,
	int* dilOffset, int dilCount, bool isOpening)
{
	int32_t imageSize = width * height * CHANNELS;
	int32_t size = imageSize * sizeof(uint8_t);
	int sizeX = width + structWidth - 1;
	int sizeY = height + structHeight - 1;
	int32_t channelSize = sizeX * sizeY * sizeof(uint8_t);
	int32_t erSize = erCount * sizeof(int);
	int32_t dilSize = dilCount * sizeof(int);
	uint8_t *d_Image, *d_output;
	uint8_t* output = (uint8_t*)malloc(size);
	//image channels
	uint8_t *d_red, *d_green, *d_blue;
	uint8_t *d_outRed, *d_outGreen, *d_outBlue;
	//structuring element offsets
	int *d_erosion, *d_dilation;
	dim3 grid_2D((sizeX + BLOCK_2D - 1) / BLOCK_2D, 
		(sizeY + BLOCK_2D - 1) / BLOCK_2D);
	dim3 block_2D(BLOCK_2D, BLOCK_2D);

	if (!output)
	{
		return NULL;
	}
	//image allocation
	hipMalloc((void**)&d_Image, size);
	hipMalloc((void**)&d_output, size);
	hipMemcpy(d_Image, image, size, hipMemcpyHostToDevice);
	//image channels allocation
	hipMalloc((void**)&d_red, channelSize);
	hipMalloc((void**)&d_green, channelSize);
	hipMalloc((void**)&d_blue, channelSize);
	hipMalloc((void**)&d_outRed, channelSize);
	hipMalloc((void**)&d_outGreen, channelSize);
	hipMalloc((void**)&d_outBlue, channelSize);
	//offset allocation
	hipMalloc((void**)&d_erosion, erSize);
	hipMemcpy(d_erosion, erOffset, erSize, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_dilation, dilSize);
	hipMemcpy(d_dilation, dilOffset, dilSize, hipMemcpyHostToDevice);
	
	if (isOpening)
	{
		//split channels
		SplitChannels<<<grid_2D, block_2D>>>(d_Image, d_red, d_green, 
			d_blue, width, height, structWidth, structHeight, WHITE);
		//it sets default values for channels and output channels
		setDefault << <grid_2D, block_2D >> > (d_outRed, d_outGreen,
			d_outBlue, sizeX, sizeY, BLACK);
		//red channel
		Erosion <<<grid_2D, block_2D>>> (d_red,
			d_outRed, width, height, structWidth,
			structHeight, d_erosion, erCount);
		Dilation<<<grid_2D, block_2D>>>(d_outRed,
			d_red, width, height, structWidth,
			structHeight, d_dilation, dilCount);
		//green channel
		Erosion <<<grid_2D, block_2D>>> (d_green,
			d_outGreen, width, height, structWidth, 
			structHeight, d_erosion, erCount);
		Dilation<<<grid_2D, block_2D>>>(d_outGreen,
			d_green, width, height, structWidth, 
			structHeight, d_dilation, dilCount);
		//blue channel
		Erosion <<<grid_2D, block_2D>>> (d_blue,
			d_outBlue, width, height, structWidth, 
			structHeight, d_erosion, erCount);
		Dilation<<<grid_2D, block_2D>>>(d_outBlue, 
			d_blue, width, height, structWidth, 
			structHeight, d_dilation, dilCount);
	}
	else
	{
		//split channels
		SplitChannels <<<grid_2D, block_2D >>> (d_Image, d_red, d_green, 
			d_blue, width, height, structWidth, structHeight, BLACK);
		//it sets default values for channels and output channels
		setDefault << <grid_2D, block_2D >> > (d_outRed, d_outGreen,
			d_outBlue, sizeX, sizeY, WHITE);
		//red channel
		Dilation <<<grid_2D, block_2D >>> (d_red,
			d_outRed, width, height, structWidth,
			structHeight, d_dilation, dilCount);
		Erosion <<<grid_2D, block_2D >>> (d_outRed,
			d_red, width, height, structWidth,
			structHeight, d_erosion, erCount);
		//green channel
		Dilation <<<grid_2D, block_2D >>> (d_green,
			d_outGreen, width, height, structWidth, 
			structHeight, d_dilation, dilCount);
		Erosion <<<grid_2D, block_2D >>> (d_outGreen,
			d_green, width, height, structWidth, 
			structHeight, d_erosion, erCount);
		//blue channel
		Dilation <<<grid_2D, block_2D >>> (d_blue,
			d_outBlue, width, height, structWidth,
			structHeight, d_dilation, dilCount);
		Erosion <<<grid_2D, block_2D >>> (d_outBlue,
			d_blue, width, height, structWidth, 
			structHeight, d_erosion, erCount);
	}
	//compose channels
	ComposeImage<<<grid_2D, block_2D>>>(d_output, 
		d_red, d_green, d_blue, structWidth,
		structHeight, width, height);
	hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);
	hipFree(d_Image);
	hipFree(d_red);
	hipFree(d_green);
	hipFree(d_blue);
	hipFree(d_outRed);
	hipFree(d_outGreen);
	hipFree(d_outBlue);
	hipFree(d_erosion);
	hipFree(d_dilation);
	return output;
}



