#include "hip/hip_runtime.h"
#include "DiamondSquareCuda.h"
#include <hiprand.h>
#include <math.h>

#define SIZE 512

/*
*	Kernel that executes the diamond step
*		matrix: the matrix that has to be computed
*		random: random values
*		currentSize: the current size of the matrix rows/columns in which 
*			apply the diamond step
*		matrixSize: the length of matrix rows/columns
*		randValue: random seed
*/
__global__ void DiamondStep(uint8_t* matrix, unsigned *random, 
	int currentSize, int matrixSize, int randValue)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int half = currentSize / 2;
	int minRand = -randValue;
	int row = y * currentSize + half;
	int col = x * currentSize + half;
	int value;
	value = (matrix[(row - half)*matrixSize + (col - half)] +
		matrix[(row - half)*matrixSize + (col + half)] +
		matrix[(row + half)*matrixSize + (col - half)] +
		matrix[(row + half)*matrixSize + (col + half)] +
		//VERSION 1
		//(random[x*gridDim.x+y] % (randValue - minRand) + minRand)) / 4;
		//VERSION 2
		(random[row*matrixSize + col] % (randValue - minRand) + minRand)) / 4;
	matrix[row*matrixSize + col] = value;
}

/*
*	Kernel that executes the square step
*		matrix: the matrix that has to be computed
*		random: random values
*		currentSize: the current size of the matrix rows/columns in which 
*			apply the diamond step
*		matrixSize: the length of matrix rows/columns
*		maxRowThread: the length of a row, using the x index
*		maxColThread: the length of a column, using the y index
*		randValue: random seed
*/
__global__ void SquareStep(uint8_t* matrix, unsigned* random, int currentSize, 
	int matrixSize, int maxRowThread, int maxColThread, int randValue)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	/*x/y can be greater than maxRowThread/maxColThread because the number
	of created threads might not be multiple of the number of threads in a block*/
	if (x < maxRowThread && y < maxColThread)
	{
		int half = currentSize / 2;
		int minRand = -randValue;
		int value = 0;
		int div = 0;
		int cond;
		int elemX = x * currentSize*(y % 2 == 0) +
			y * half*(y % 2 != 0);
		int elemY = (y*half + half)*(y % 2 == 0) +
			x * currentSize*(y % 2 != 0);
		// CUDA VERSION 2: it uses conditions as variables 
		// to avoid divergent branches
		cond = elemX != 0;
		value += matrix[(elemX - half * cond) *
			matrixSize + elemY] * cond;
		div += cond;
		cond = elemX != matrixSize - 1;
		value += matrix[(elemX + half * cond) *
			matrixSize + elemY] * cond;
		div += cond;
		cond = elemY != 0;
		value += matrix[elemX * matrixSize + 
			elemY - half * cond] * cond;
		div += cond;
		cond = elemY != matrixSize - 1;
		value += matrix[elemX*matrixSize + elemY 
			+ half * cond] * cond;
		div += cond;
		/*
		// CUDA VERSION 1: it uses divergent branches
		if (elemX != 0)
		{
			value += matrix[(elemX - half)*matrixSize + elemY];
			div++;
		}
		if (elemX != matrixSize-1)
		{
			value += matrix[(elemX + half)*matrixSize + elemY];
			div++;
		}
		if (elemY != 0)
		{
			value += matrix[elemX*matrixSize + elemY - half];
			div++;
		}
		if (elemY != matrixSize-1)
		{
			value += matrix[elemX*matrixSize + elemY + half];
			div++;
		}*/
		//VERSION 1: random index is correct for the 
		//first version of random generation but not for the second one
		//value += (minRand + random[x*gridDim.x+y] % (randValue - minRand));
		//VERSION 2
		value += (minRand + random[elemX*matrixSize+elemY] % (randValue - minRand));
		matrix[elemX*matrixSize + elemY] = value / div;
	}
}

/*
*	Function that will be called by the Unreal Engine classes to execute
*	the diamond-square algorithm with CUDA
*		matrix: the matrix that has to be computed
*		matrixSize: the length of rows/columns of the matrix that has to be computed
*		randomValue: the random seed at the beginning of the execution
*/
void CudaAlgorithm::CudaDiamondSquare(uint8_t* matrix, 
	int matrixSize, int randomValue)
{
	uint8_t *d_matrix;
	unsigned *d_random;
	int size = matrixSize * matrixSize * sizeof(uint8_t);
	int last = matrixSize - 1;
	int half; 
	int count = 0;
	int diamondSize = CudaAlgorithm::PowerInt(2, count);
	int squareRowSize = 0;
	int squareColSize = 0;
	int randValue = randomValue;
	hiprandGenerator_t generator;
	//Allocation of device pointer
	hipMalloc((void**)&d_matrix, size);
	//Copy of the values from host pointer to the device
	hipMemcpy(d_matrix, matrix, size, hipMemcpyHostToDevice);
	//VERSION 2
	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetGeneratorOrdering(generator, HIPRAND_ORDERING_PSEUDO_SEEDED);
	hipMalloc((void**)&d_random, matrixSize*matrixSize);
	hiprandGenerate(generator, d_random, matrixSize*matrixSize);

	while (last > 1)
	{
		int d_length = CudaAlgorithm::PowerInt(2, count);
		int S_row, S_column;
		half = last / 2;
		randValue = randValue / 2 > 1 ? randValue / 2 : 1;
		//It checks if the number of threads is less than the maximum
		if (pow(d_length, 2) < SIZE)
		{
			diamondSize = d_length;
		}
		if (last != matrixSize - 1)
		{
			// VERSION 1
			//hiprandDestroyGenerator(generator);
			hipFree(d_random);
		}
		//hipMalloc((void**)&d_random, sizeof(unsigned)*d_length*d_length);
		dim3 D_block(diamondSize, diamondSize);
		dim3 D_grid(d_length / diamondSize, d_length / diamondSize);
		//VERSION 1
		//Creation of the generator of random numbers
		//hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
		//It creates random numbers
		//hiprandGenerate(generator, d_random, d_length*d_length);
		DiamondStep <<<D_grid, D_block>>> (d_matrix, d_random, last,
			matrixSize, randValue);
		count++;
		S_row = CudaAlgorithm::PowerInt(2, count-1)+1;
		S_column = CudaAlgorithm::PowerInt(2, count);
		if (S_row*S_column < SIZE)
		{
			squareRowSize = S_row;
			squareColSize = S_column;
		}
		//VERSION 1
		//hipFree(d_random);
		//hiprandDestroyGenerator(generator);
		hipMalloc((void**)&d_random, sizeof(int)*S_row*S_column);
		dim3 S_block(squareRowSize, squareColSize);
		dim3 S_grid((S_row + squareRowSize - 1) / squareRowSize, 
			S_column / squareColSize);
		//VERSION 1: It creates other random numbers for square step
		//hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
		//hiprandGenerate(generator, (unsigned*)d_random, S_row*S_column);
		SquareStep <<<S_grid, S_block>>> (d_matrix, d_random, last, matrixSize,
			S_row, S_column, randValue);
		last = half;
	}
	//Copy of the values from device pointes to host
	hipMemcpy(matrix, d_matrix, size, hipMemcpyDeviceToHost);
	//It frees the allocated device memory
	hipFree(d_matrix);
	hiprandDestroyGenerator(generator);
	hipFree(d_random);
}

/*
*	Private function used to calculate the power of integer values
*	because the pow function uses double values and creates warnings
*	when we try to cast them to integer.
*		base: the base of the power
*		exp: the exponent
*/
int CudaAlgorithm::PowerInt(int base, int exp)
{
	int power = 1;
	for (int i = 1; i <= exp; i++)
	{
		power *= base;
	}
	return power;
}
